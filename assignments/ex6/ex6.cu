
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 8192
#define THREAD_PER_BLOCK_SIDE 32
#define THREAD_PER_BLOCK THREAD_PER_BLOCK_SIDE*THREAD_PER_BLOCK_SIDE

__global__ void transpose(int * in, int * out, int size)
{
    //int temp_side = THREAD_PER_BLOCK;
    __shared__ int temp_matrix[THREAD_PER_BLOCK_SIDE][THREAD_PER_BLOCK_SIDE];

    //int temp_i = threadIdx.y*temp_side + threadIdx.x;
    //int temp_i_t = threadIdx.x*temp_side + threadIdx.y;
    int global_i = blockIdx.y*blockDim.y*size + blockIdx.x*blockDim.x + threadIdx.y*size + threadIdx.x;
    int global_i_t = blockIdx.x*blockDim.y*size + blockIdx.y*blockDim.x + threadIdx.y*size + threadIdx.x;

    // copy submatrix (transposed) in shared memory
    temp_matrix[threadIdx.x][threadIdx.y] = in[global_i_t];

    __syncthreads();

    // copy submatrix in main memory
    out[global_i] = temp_matrix[threadIdx.y][threadIdx.x];

}

int correct(int* a, int* b, int size)
{   
    int i;
    for(i=0; i<size; i++)
        if(a[i]!=b[(i%size)*size + i/size]) return 0;
    return 1;
}

int main()
{

    int * h_in, * h_out;
    int * d_in, * d_out;
    int size = N*N;
    int size_in_memory = size * sizeof(int);
    int i;


    // timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    //allocate memory in host and device
    h_in = (int *)malloc(size_in_memory);
    h_out = (int *)malloc(size_in_memory);

    hipMalloc((void**)&d_in, size_in_memory);
    hipMalloc((void**)&d_out, size_in_memory);


    //fill matrix in host
    for(i = 0; i<size; i++)
        h_in[i] = i;


    //transfer matrix from host to device
    hipMemcpy(d_in, h_in, size_in_memory, hipMemcpyHostToDevice);


    //transpose matrix in device
    dim3 grid, block;
    block.x = THREAD_PER_BLOCK_SIDE;
    block.y = THREAD_PER_BLOCK_SIDE;
    grid.x = N / block.x;
    grid.y = N / block.y;

    hipEventRecord(start);
    transpose<<< grid, block >>>(d_in, d_out, N);
    hipEventRecord(stop);


    //transfer matrix in host
    hipMemcpy(h_out, d_out, size_in_memory, hipMemcpyDeviceToHost);


    //free memory   
    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);


    // correctness test
    printf("\ncorrecteness: %d \n", correct(d_in, d_out, size));
   

    //showing Bandwidth
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("\nEffective Bandwidth (GB/s): %f \n", size_in_memory/milliseconds/1e6);


    return 0;
}
